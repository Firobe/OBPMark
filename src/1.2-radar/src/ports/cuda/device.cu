#include "hip/hip_runtime.h"
/**
 * \file device.c
 * \brief Benchmark #1.2 GPU version (cuda) device initialization. 
 * \author Marc Sole Bonet (BSC)
 */
#include "device.h"
#include "processing.h"

uint32_t next_power_of_two(uint32_t n)
{
    uint32_t v = n;
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

void init(
	radar_data_t *radar_data,
	radar_time_t *t,
	char *device_name
	)
{
    init(radar_data,t, 0,0, device_name);
}


void init(
	radar_data_t *radar_data,
	radar_time_t *t,
	int platform,
	int device,
	char *device_name
	)
{
    hipSetDevice(device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    strcpy(device_name,prop.name);
    //event create 
    t->start = new hipEvent_t;
    t->stop = new hipEvent_t;
    t->start_memory_copy_device = new hipEvent_t;
    t->stop_memory_copy_device = new hipEvent_t;
    t->start_memory_copy_host = new hipEvent_t;
    t->stop_memory_copy_host= new hipEvent_t;
    
    hipEventCreate(t->start);
    hipEventCreate(t->stop);
    hipEventCreate(t->start_memory_copy_device);
    hipEventCreate(t->stop_memory_copy_device);
    hipEventCreate(t->start_memory_copy_host);
    hipEventCreate(t->stop_memory_copy_host);

}


bool device_memory_init(
	radar_data_t *radar_data,
	radar_params_t *params,
    unsigned int out_height,
    unsigned int out_width
	)
{	
    unsigned int patch_width = params->rsize<<1;
    unsigned int patch_extended_width = next_power_of_two(params->rsize)<<1;
    unsigned int patch_height = params->apatch;

    radar_data->out_height = out_height;
    radar_data->out_width = out_width;
    radar_data->host_params = params;

    /* FFT plans */
    hipfftPlan1d(&radar_data->rrf_plan, next_power_of_two(params->rsize), HIPFFT_C2C, 1);
    hipfftPlan1d(&radar_data->arf_plan, params->apatch, HIPFFT_C2C, 1);
    hipfftPlan1d(&radar_data->range_plan, next_power_of_two(params->rsize), HIPFFT_C2C, params->apatch * params->npatch);
    hipfftPlan1d(&radar_data->azimuth_plan, params->apatch, HIPFFT_C2C, params->rvalid * params->npatch);

    hipError_t err = hipSuccess;

//	/* radar_data_t memory allocation */
//	//RANGE & AZIMUTH DATA
	err = hipMalloc((void **)&(radar_data->range_data),
	        sizeof(float) * params->npatch * patch_height * patch_extended_width);
    if (err != hipSuccess) {printf("rangedata\n"); return false;}
    err = hipMemset(radar_data->range_data, 0,
	        sizeof(float) * params->npatch * patch_height * patch_extended_width);
    if (err != hipSuccess) return false;

	err = hipMalloc((void **)&(radar_data->azimuth_data),
	        sizeof(float) * params->npatch * patch_height * patch_width);
    if (err != hipSuccess) {printf("azi data\n"); return false;}
    err = hipMemset(radar_data->azimuth_data, 0,
	        sizeof(float) * params->npatch * patch_height * patch_width);
    if (err != hipSuccess) return false;

  	//MULTILOOK DATA
	err = hipMalloc((void **)&(radar_data->ml_data),
	        sizeof(float) * params->npatch * out_height * out_width);
    if (err != hipSuccess) {printf("ml data\n"); return false;}

  	//OUTPUT DATA
	err = hipMalloc((void **)&(radar_data->output_image),
	        sizeof(uint8_t) * params->npatch * out_height * out_width);
    if (err != hipSuccess) {printf("out data\n"); return false;}

  	//PARAMS
	err = hipMalloc((void **)&(radar_data->params), sizeof(radar_params_t));
    if (err != hipSuccess) {printf("params data\n"); return false;}
//    
    //RANGE REF. FUNCTION
	err = hipMalloc((void **)&(radar_data->rrf), sizeof(float) * patch_extended_width);
    if (err != hipSuccess) {printf("rrf data\n"); return false;}
    err = hipMemset(radar_data->rrf, 0, sizeof(float) * patch_extended_width);
    if (err != hipSuccess) return false;

	//AZIMUTH REF. FUNCTION
	err = hipMalloc((void **)&(radar_data->arf), sizeof(float) * (patch_height<<1));
    if (err != hipSuccess) {printf("arf data\n"); return false;}
    err = hipMemset(radar_data->arf, 0, sizeof(float) * (patch_height<<1));
    if (err != hipSuccess) return false;

    //DOPPLER CENTROID VALUE
//	err = hipMalloc((void **)&(radar_data->fDc), sizeof(float));
//    if (err != hipSuccess) return false;
//    err = hipMemset(radar_data->fDc, 0, sizeof(float));
//    if (err != hipSuccess) return false;

   // //DOPPLER AUXILIAR BUFFER
   // err = hipMalloc((void **)&(radar_data->aux), sizeof(float) * patch_width);
   // if (err != hipSuccess) return false;
   // hipMemset(radar_data->aux, 0, sizeof(float) * patch_width);
   // if (err != hipSuccess) return false;
//
    //RCMC TABLE
	err = hipMalloc((void **)&(radar_data->offsets), sizeof(uint32_t) * params->rvalid * patch_height);
    if (err != hipSuccess) {printf("off data\n"); return false;}

//    //Device pointer
//	err = hipMalloc((void **)&(radar_data->gpu_ptr), sizeof(radar_data_t));
//    if (err != hipSuccess) return false;

    return true;
}

void copy_memory_to_device(
	radar_data_t *radar_data,
	radar_time_t *t,
	framefp_t *input_data,
	radar_params_t *input_params
	)
{
    hipEventRecord(*t->start_memory_copy_device);

    /* Copy params */
    hipMemcpy(radar_data->params, input_params, sizeof(radar_params_t), hipMemcpyHostToDevice);
    uint32_t width = input_params->rsize<<1;
    uint32_t height = input_params->apatch;
    uint32_t line_width = next_power_of_two(width);
    uint32_t patch_size = line_width * height; // * sizeof(float);
    for (uint32_t i = 0; i < input_params->npatch; i++ )
        for(uint32_t j = 0; j < height; j++){
            uint32_t offs = i * patch_size + j * line_width;
            hipMemcpy(&radar_data->range_data[offs], &input_data[i].f[j * width], width * sizeof(float), hipMemcpyHostToDevice);
        }

    //hipMemcpy(radar_data->gpu_ptr, radar_data, sizeof(radar_data_t), hipMemcpyHostToDevice);

    hipEventRecord(*t->stop_memory_copy_device);
}


void process_benchmark(
	radar_data_t *radar_data,
	radar_time_t *t
	)
{    
    hipDeviceSynchronize();
    hipEventRecord(*t->start);

    radar_params_t *params = radar_data->host_params;

    /* SAR RANGE REFERENCE */
    int n_blocks = (params->rsize-1)/BLOCK_SIZE+1;
    // compute reference function
    uint32_t nit = floor(params->tau * params->fs);
    SAR_range_ref<<<n_blocks,BLOCK_SIZE>>>(radar_data->rrf, radar_data->params, nit);
    // perform fft
    hipfftExecC2C(radar_data->rrf_plan, (hipfftComplex*) radar_data->rrf, (hipfftComplex*) radar_data->rrf, HIPFFT_FORWARD);


    /* DOPPLER CENTROID */
    dim3 blockSize(TILE_SIZE,TILE_SIZE,1);
    float const_k = params->PRF/(2*pi*params->rsize);
    SAR_DCE<<<params->rsize,BLOCK_SIZE,sizeof(float)*2*params->apatch>>>(radar_data->range_data, radar_data->params, const_k);

//    printffDc<<<1,1>>>();
    /* RCMC table */
    dim3 gridSize(params->apatch/TILE_SIZE,(params->rvalid-1)/TILE_SIZE+1,1);
    SAR_rcmc_table<<<gridSize, blockSize>>>(radar_data->params, radar_data->offsets);

    /* SAR AZIMUTH REFERENCE */
    // compute azimuth values
    n_blocks = (params->apatch)/BLOCK_SIZE;
    // Compute azimuth reference
    SAR_azimuth_ref<<<n_blocks, BLOCK_SIZE>>>(radar_data->arf, radar_data->params);
    // perform fft
    hipfftExecC2C(radar_data->arf_plan, (hipfftComplex*) radar_data->arf, (hipfftComplex*) radar_data->arf, HIPFFT_FORWARD);

    /* Begin patch processing */
    //SAR Range Compress
    hipfftExecC2C(radar_data->range_plan, (hipfftComplex*) radar_data->range_data, (hipfftComplex*) radar_data->range_data, HIPFFT_FORWARD);
    gridSize = {next_power_of_two(params->rsize)/TILE_SIZE, params->apatch/TILE_SIZE, params->npatch};
    SAR_ref_product<<<gridSize,blockSize>>>(radar_data->range_data, radar_data->rrf, next_power_of_two(params->rsize), params->apatch);
    hipfftExecC2C(radar_data->range_plan, (hipfftComplex*) radar_data->range_data, (hipfftComplex*) radar_data->range_data, HIPFFT_BACKWARD);
    //after IFFT data needs to be idvided by next_power_of_two(rsize), we do that when transposing
    SAR_transpose<<<gridSize, blockSize>>>(radar_data->range_data, radar_data->azimuth_data, next_power_of_two(params->rsize), params->apatch, params->apatch, params->rvalid);
    hipfftExecC2C(radar_data->azimuth_plan, (hipfftComplex*) radar_data->azimuth_data, (hipfftComplex*) radar_data->azimuth_data, HIPFFT_FORWARD);

    gridSize= {params->apatch/TILE_SIZE,(params->rvalid-1)/TILE_SIZE+1,params->npatch};
    /* RCMC */
    SAR_rcmc<<<gridSize,blockSize>>>(radar_data->azimuth_data , radar_data->offsets, params->apatch, params->rvalid);

    /* Azimuth Compress */
    SAR_ref_product<<<gridSize, blockSize>>>(radar_data->azimuth_data, radar_data->arf, params->apatch, params->rvalid);
    hipfftExecC2C(radar_data->azimuth_plan, (hipfftComplex*) radar_data->azimuth_data, (hipfftComplex*) radar_data->azimuth_data, HIPFFT_BACKWARD);
    //after IFFT data needs to be idvided by next_power_of_two(rsize), we do that when transposing
    SAR_transpose<<<gridSize, blockSize>>>(radar_data->azimuth_data, radar_data->range_data, params->apatch, next_power_of_two(params->rsize), params->rvalid, params->apatch);

    gridSize= {(radar_data->out_width-1)/TILE_SIZE+1,(radar_data->out_height-1)/TILE_SIZE+1,1};
    SAR_multilook<<<gridSize,blockSize>>>(radar_data->range_data, radar_data->ml_data, radar_data->params, radar_data->out_width, radar_data->out_height);
    quantize<<<gridSize,blockSize>>>(radar_data->ml_data, radar_data->output_image, radar_data->out_width, radar_data->out_height);

    hipEventRecord(*t->stop);

}

void copy_memory_to_host(
	radar_data_t *radar_data,
	radar_time_t *t,
	frame8_t *output_radar
	)
{
    hipEventRecord(*t->start_memory_copy_host);
    uint32_t  width = output_radar->w;
    uint32_t  height = output_radar->h;
    hipMemcpy(output_radar->f, radar_data->output_image, sizeof(uint8_t) * width * height, hipMemcpyDeviceToHost);
    hipEventRecord(*t->stop_memory_copy_host);
}


void get_elapsed_time(
	radar_data_t *radar_data, 
	radar_time_t *t, 
	bool csv_format,
	bool database_format,
	bool verbose_print,
	long int timestamp
	)
{	
    hipEventSynchronize(*t->stop_memory_copy_host);
    float milliseconds_h_d = 0, milliseconds = 0, milliseconds_d_h = 0;
    // memory transfer time host-device
    hipEventElapsedTime(&milliseconds_h_d, *t->start_memory_copy_device, *t->stop_memory_copy_device);
    // kernel time
    hipEventElapsedTime(&milliseconds, *t->start, *t->stop);
    //  memory transfer time device-host
    hipEventElapsedTime(&milliseconds_d_h, *t->start_memory_copy_host, *t->stop_memory_copy_host);

	if (csv_format)
	{
		printf("%.10f;%.10f;%.10f;\n", milliseconds_h_d, milliseconds, milliseconds_d_h);
	}
	else if (database_format)
	{
		printf("%.10f;%.10f;%.10f;%ld;\n", milliseconds_h_d, milliseconds, milliseconds_d_h, timestamp);
	}
	else if(verbose_print)
	{
		printf("Elapsed time Host->Device: %.10f milliseconds\n", milliseconds_h_d);
		printf("Elapsed time kernel: %.10f milliseconds\n", milliseconds );
		printf("Elapsed time Device->Host: %.10f milliseconds\n", milliseconds_d_h);
	}
}


void clean(
	radar_data_t *radar_data,
	radar_time_t *t
	)
{

	/* Clean time */
	free(t);

    hipError_t err = hipSuccess;

	/* Clean radar data */
	err = hipFree(radar_data->range_data);
	if(err != hipSuccess) { fprintf(stderr, "Failed to free device data (error code %s)!\n", hipGetErrorString(err)); return; }

	err = hipFree(radar_data->azimuth_data);
	if(err != hipSuccess) { fprintf(stderr, "Failed to free device data (error code %s)!\n", hipGetErrorString(err)); return; }

	err = hipFree(radar_data->ml_data);
	if(err != hipSuccess) { fprintf(stderr, "Failed to free device data (error code %s)!\n", hipGetErrorString(err)); return; }

	err = hipFree(radar_data->output_image);
	if(err != hipSuccess) { fprintf(stderr, "Failed to free device data (error code %s)!\n", hipGetErrorString(err)); return; }

	err = hipFree(radar_data->params);
	if(err != hipSuccess) { fprintf(stderr, "Failed to free device data (error code %s)!\n", hipGetErrorString(err)); return; }

//	err = hipFree(radar_data->aux);
//	if(err != hipSuccess) { fprintf(stderr, "Failed to free device data (error code %s)!\n", hipGetErrorString(err)); return; }

//	err = hipFree(radar_data->fDc);
//	if(err != hipSuccess) { fprintf(stderr, "Failed to free device data (error code %s)!\n", hipGetErrorString(err)); return; }

	err = hipFree(radar_data->rrf);
	if(err != hipSuccess) { fprintf(stderr, "Failed to free device data (error code %s)!\n", hipGetErrorString(err)); return; }

	err = hipFree(radar_data->arf);
	if(err != hipSuccess) { fprintf(stderr, "Failed to free device data (error code %s)!\n", hipGetErrorString(err)); return; }

	err = hipFree(radar_data->offsets);
	if(err != hipSuccess) { fprintf(stderr, "Failed to free device data (error code %s)!\n", hipGetErrorString(err)); return; }

//	err = hipFree(radar_data->gpu_ptr);
//	if(err != hipSuccess) { fprintf(stderr, "Failed to free device data (error code %s)!\n", hipGetErrorString(err)); return; }

    free(radar_data);
}
